#include "hip/hip_runtime.h"
#pragma once

#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>
#include "cudasolarsystem.cuh"


                                       
__device__ __constant__ double kMSun = 1988470000000000000000000000000.0; // 1.98847 * pow(10.0, 30);
__device__ __constant__ double kG = 0.0000000000667543; //6.67543 * pow(10.0, -11);
__device__ __constant__ double kAU = 149600000000.0; // 1.496 * pow(10.0, 11);         // m^3
const int kMaxNumberAsteroid = 200;
__device__ __constant__ int kMaxNumberAsteroidDev = kMaxNumberAsteroid;
const int kNumberThreadPerBlock = 1024;



__global__ void addKernel2(double* a) {
  int i = threadIdx.x;
  a[i] *= -1;;
}

                                    
// m1, mSun, �d?, ->F
__global__ void forceKernel(float* m, float* fx, float* fy, float* fz, 
                            double* x, double* y, double* z, const int size,
                            const int* asteroidID, const int* numberAsteroid){
  
  int i = threadIdx.x;
  int j = blockIdx.x;
  i = i + j * blockDim.x;
  if (size > i) {

    // Sun force
    double distance = x[i] * x[i] + y[i] * y[i] + z[i] * z[i];
    if (distance == 0) { distance = 0.01f; }
    float moduleF= 1.0f/-sqrt(distance);
    if (distance <= 100) { moduleF *= -1.0f; }
    float dirX = x[i] * moduleF;
    float dirY = y[i] * moduleF;
    float dirZ = z[i] * moduleF;

    float force = kG * ((kMSun * m[i]) / distance);

    fx[i] = force * dirX * 10000000.0f;
    fy[i] = force * dirY * 10000000.0f;
    fz[i] = force * dirZ * 10000000.0f;


    // other Asteroids
    int n = numberAsteroid[i];
    int offsetData = kMaxNumberAsteroidDev * i;
    for (int a = 0; a < n; ++a) {

      int asteroid = asteroidID[offsetData + a];
      float tmpX = x[asteroid] - x[i];
      float tmpY = y[asteroid] - y[i];
      float tmpZ = z[asteroid] - z[i];


      distance = tmpX * tmpX + tmpY * tmpY + tmpZ * tmpZ;
      if (distance == 0) { distance = 0.01f; }
      moduleF = 1/sqrt(distance);
      if (distance <= 100) { moduleF *= -1.0f; }

      dirX = tmpX * moduleF;
      dirY = tmpY * moduleF;
      dirZ = tmpZ * moduleF;

      force = kG * ((m[i] * m[asteroid]) / distance);


      fx[i] += force * dirX;
      fy[i] += force * dirY;
      fz[i] += force * dirZ;

    }
  }

}

// m1, F, ->A
__global__ void accelerationKernel(float* m, float *fx, float* fy, float* fz, 
                                   float *ax, float* ay, float* az, const int size){
  int i = threadIdx.x;
  int j = blockIdx.x;
  i = i + j * blockDim.x;
  float mInv = 1 / m[i];
  if (size > i) {
    ax[i] = fx[i] *mInv;
    ay[i] = fy[i] *mInv;
    az[i] = fz[i] *mInv;
  }
}

// V, A, t
__global__ void velocityKernel(float* vx, float* vy, float* vz, 
                               float* ax, float* ay, float* az,
                               float* t, const int size){
  int i = threadIdx.x;
  int j = blockIdx.x;
  i = i + j * blockDim.x;
  if (size > i) {
    float velX = vx[i] + ax[i] * t[0];
    float velY = vy[i] + ay[i] * t[0];
    float velZ = vz[i] + az[i] * t[0];

    vx[i] = velX;
    vy[i] = velY;
    vz[i] = velZ;
  }
}

// P, V, A, t
__global__ void positionKernel(double* x, double* y, double* z, float* vx, 
                               float* vy, float* vz, float* ax, float* ay, 
                               float* az, float* t, const int size){
  int i = threadIdx.x;
  int j = blockIdx.x;
  i = i + j * blockDim.x;
  if (size > i) {
    float time = t[0];

    float pX = x[i] + vx[i] * time + (ax[i] * time * time * 0.5f);
    float pY = y[i] + vy[i] * time + (ay[i] * time * time * 0.5f);
    float pZ = z[i] + vz[i] * time + (az[i] * time * time * 0.5f);

    x[i] = pX;
    y[i] = pY;
    z[i] = pZ;
  }

}


hipError_t initAsteroidBeltCuda(AsteroidBeltDeviceData& asteroidDataGPU, int size) {


  std::chrono::steady_clock::time_point start, end;
  std::chrono::microseconds duration;
  start = std::chrono::high_resolution_clock::now();

  hipError_t cudaStatus = hipSetDevice(0);
  if (hipSuccess != cudaStatus) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }


  //Allocation of the device memory
  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_t, sizeof(float));
  if (hipSuccess != cudaStatus) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_m, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_x, size * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_y, size * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_z, size * sizeof(double));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_vx, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_vy, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_vz, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_ax, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_ay, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_az, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_fx, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_fy, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_fz, size * sizeof(float));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_outputData, size * kMaxNumberAsteroid * sizeof(int));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }

  cudaStatus = hipMalloc((void**)&asteroidDataGPU.dev_outputSize, size * sizeof(int));
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMalloc failed!");
    goto Error;
  }



Error:
  end = std::chrono::high_resolution_clock::now();
  duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  printf("Allocation CudaMemory: %f microseconds. \n", duration.count() / 1000000.0f);


  return cudaStatus;

}



hipError_t updateAsteroidBeltCuda(double* x, double* y, double* z,
                                   float* vx, float* vy, float* vz,
                                   float* m,  float* t, int size,
                                   int* outputData, int* outputSize,
                                   AsteroidBeltDeviceData* asteroidDataGPU, MeasureTime* chrono,
                                  int id)
{
  

  chrono->startUpdate[id] = std::chrono::high_resolution_clock::now();

  hipError_t cudaStatus;
  int numberBlocks = (int)ceilf((float)size / kNumberThreadPerBlock);
  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaStatus = hipSetDevice(0);
  if (hipSuccess != cudaStatus) {
    fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    goto Error;
  }

  
  // Copy input vectors from host memory to GPU buffers.
  cudaStatus = hipMemcpy(asteroidDataGPU->dev_t, t, sizeof(float), hipMemcpyHostToDevice);
  if (hipSuccess != cudaStatus) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_m, m, size * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_x, x, size * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_y, y, size * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_z, z, size * sizeof(double), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_vx, vx, size * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_vy, vy, size * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_vz, vz, size * sizeof(float), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(asteroidDataGPU->dev_outputData, outputData, size * kMaxNumberAsteroid *  sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }
  cudaStatus = hipMemcpy(asteroidDataGPU->dev_outputSize, outputSize, size * sizeof(int), hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  chrono->startF[id] = std::chrono::high_resolution_clock::now();

  
  
  //Force Kernel
  {
    // Launch a kernel on the GPU with one thread for each element.
    forceKernel << <numberBlocks, kNumberThreadPerBlock >> > (asteroidDataGPU->dev_m, 
    asteroidDataGPU->dev_fx, asteroidDataGPU->dev_fy, asteroidDataGPU->dev_fz,
    asteroidDataGPU->dev_x, asteroidDataGPU->dev_y, asteroidDataGPU->dev_z, size,
    asteroidDataGPU->dev_outputData, asteroidDataGPU->dev_outputSize);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "forceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching forceKernel!\n", cudaStatus);
      goto Error;
    }
    
  }

  chrono->startA[id] = std::chrono::high_resolution_clock::now();
  
  //Acceleration Kernel
  {
    // Launch a kernel on the GPU with one thread for each element.
    accelerationKernel << <numberBlocks, kNumberThreadPerBlock >> > (asteroidDataGPU->dev_m, 
      asteroidDataGPU->dev_fx, asteroidDataGPU->dev_fy, asteroidDataGPU->dev_fz, 
      asteroidDataGPU->dev_ax, asteroidDataGPU->dev_ay, asteroidDataGPU->dev_az, size);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "accelerationKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching accelerationKernel!\n", cudaStatus);
      goto Error;
    }
  }

  chrono->startV[id] = std::chrono::high_resolution_clock::now();
  //Velocity Kernel
  {
    // Launch a kernel on the GPU with one thread for each element.
    velocityKernel << <numberBlocks, kNumberThreadPerBlock >> > (
      asteroidDataGPU->dev_vx, asteroidDataGPU->dev_vy, asteroidDataGPU->dev_vz, 
      asteroidDataGPU->dev_ax, asteroidDataGPU->dev_ay, asteroidDataGPU->dev_az, 
      asteroidDataGPU->dev_t, size);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "velocityKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching velocityKernel!\n", cudaStatus);
      goto Error;
    }
  }
  chrono->startP[id] = std::chrono::high_resolution_clock::now();

  //Position Kernel
  {
    // Launch a kernel on the GPU with one thread for each element.
    positionKernel << <numberBlocks, kNumberThreadPerBlock >> > (
      asteroidDataGPU->dev_x, asteroidDataGPU->dev_y, asteroidDataGPU->dev_z,
      asteroidDataGPU->dev_vx, asteroidDataGPU->dev_vy, asteroidDataGPU->dev_vz, 
      asteroidDataGPU->dev_ax, asteroidDataGPU->dev_ay, asteroidDataGPU->dev_az, 
      asteroidDataGPU->dev_t, size);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "positionKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
      goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (hipSuccess != cudaStatus) {
      fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching positionKernel!\n", cudaStatus);
      goto Error;
    }
  }
  chrono->endP[id] = std::chrono::high_resolution_clock::now();

  

  //Copy back the information from GPU buffer to Host
  cudaStatus = hipMemcpy(t, asteroidDataGPU->dev_t, sizeof(float), hipMemcpyDeviceToHost);
  if (hipSuccess != cudaStatus) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(x, asteroidDataGPU->dev_x, size * sizeof(double), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(y, asteroidDataGPU->dev_y, size * sizeof(double), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(z, asteroidDataGPU->dev_z, size * sizeof(double), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(vx, asteroidDataGPU->dev_vx, size * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(vy, asteroidDataGPU->dev_vy, size * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  cudaStatus = hipMemcpy(vz, asteroidDataGPU->dev_vz, size * sizeof(float), hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!");
    goto Error;
  }

  chrono->endUpdate[id] = std::chrono::high_resolution_clock::now();
 
Error:


  return cudaStatus;
}



void endAsteroidBeltCuda(AsteroidBeltDeviceData& asteroidDataGPU) {

  hipFree(asteroidDataGPU.dev_t);
  hipFree(asteroidDataGPU.dev_m);

  hipFree(asteroidDataGPU.dev_x);
  hipFree(asteroidDataGPU.dev_y);
  hipFree(asteroidDataGPU.dev_z);

  hipFree(asteroidDataGPU.dev_vx);
  hipFree(asteroidDataGPU.dev_vy);
  hipFree(asteroidDataGPU.dev_vz);

  hipFree(asteroidDataGPU.dev_ax);
  hipFree(asteroidDataGPU.dev_ay);
  hipFree(asteroidDataGPU.dev_az);

  hipFree(asteroidDataGPU.dev_fx);
  hipFree(asteroidDataGPU.dev_fy);
  hipFree(asteroidDataGPU.dev_fz);

  hipFree(asteroidDataGPU.dev_outputData);
  hipFree(asteroidDataGPU.dev_outputSize);
}


